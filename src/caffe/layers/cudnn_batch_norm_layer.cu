
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <cfloat>
#include <vector>
//#include <math>

#include "thrust/device_vector.h"

#include "caffe/layers/cudnn_batch_norm_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void CuDNNBatchNormLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* scale_data = this->blobs_[0]->gpu_data();
  const Dtype* bias_data = this->blobs_[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* save_mean = save_mean_.mutable_gpu_data();
  Dtype* save_inv_var = save_inv_var_.mutable_gpu_data();
  double epsilon = max(this->eps_, CUDNN_BN_MIN_EPSILON);
  if (this->phase_ == TRAIN) {
    int iter_size = Caffe::iter_size();
    Dtype fraction = 1 - this->moving_average_fraction_;
    if(iter_size > 1)
      for(int i = 1;i<iter_size;i++)
        fraction = sqrt(fraction);
    // Call Batch normalization forward
    CUDNN_CHECK(cudnnBatchNormalizationForwardTraining(
      this->handle_, mode_,
      cudnn::dataType<Dtype>::one, cudnn::dataType<Dtype>::zero,
      bottom_desc_, bottom_data,
      bottom_desc_, top_data,
      scale_bias_mean_var_desc_, scale_data, bias_data,
      //1 - this->moving_average_fraction_,
      fraction,
      this->blobs_[2]->mutable_gpu_data(),  // mean
      this->blobs_[3]->mutable_gpu_data(),  // variance
      epsilon, save_mean, save_inv_var));
  } else if (this->phase_ == TEST) {
    CUDNN_CHECK(cudnnBatchNormalizationForwardInference(
      this->handle_, mode_,
      cudnn::dataType<Dtype>::one, cudnn::dataType<Dtype>::zero,
      bottom_desc_, bottom_data,
      bottom_desc_, top_data,
      scale_bias_mean_var_desc_, scale_data, bias_data,
      this->blobs_[2]->gpu_data(),  // mean
      this->blobs_[3]->gpu_data(),  // variance
      epsilon));
  } else {
    LOG(FATAL) << "Unknown phase";
  }
  // copy data to diff,send to ps
  caffe_copy(this->blobs_[2]->count(),
	this->blobs_[2]->gpu_data(),
	this->blobs_[2]->mutable_gpu_diff());
  caffe_copy(this->blobs_[3]->count(),
	this->blobs_[3]->gpu_data(),
	this->blobs_[3]->mutable_gpu_diff());
}

template <typename Dtype>
void CuDNNBatchNormLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* save_mean = save_mean_.gpu_data();
  const Dtype* save_inv_var = save_inv_var_.gpu_data();

  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const Dtype* scale_data = this->blobs_[0]->gpu_data();
  Dtype* scale_diff = this->blobs_[0]->mutable_gpu_diff();
  Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();

  double epsilon = max(this->eps_, CUDNN_BN_MIN_EPSILON);

  // call Batch Normalization Backward
  CUDNN_CHECK(cudnnBatchNormalizationBackward(
      this->handle_, mode_,
      cudnn::dataType<Dtype>::one, cudnn::dataType<Dtype>::zero,
#if CUDNN_VERSION >= 4005
      cudnn::dataType<Dtype>::one, cudnn::dataType<Dtype>::one,
#endif
      bottom_desc_, bottom_data,
      bottom_desc_, top_diff,
      bottom_desc_, bottom_diff,
      scale_bias_mean_var_desc_,
      scale_data, scale_diff, bias_diff,
      epsilon, save_mean, save_inv_var));
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNBatchNormLayer);

}  // namespace caffe

#endif
